#include <hip/hip_runtime.h>
#include <stdio.h>

//////////////////////demo1 //////////////////////////
/*
demo1 主要为了展示查看静态和动态共享变量的地址
 */
const size_t static_shared_memory_num_element = 6 * 1024; // 6KB
__shared__ char static_shared_memory[static_shared_memory_num_element];
__shared__ char static_shared_memory2[2];

__global__ void demo1_kernel()
{
    extern __shared__ char dynamic_shared_memory[]; // 静态共享变量和动态共享变量在kernel函数内/外定义都行，没有限制
    extern __shared__ char dynamic_shared_memory2[];
    printf("static_shared_memory = %p\n", static_shared_memory); // 静态共享变量，定义几个地址随之叠加
    printf("static_shared_memory2 = %p\n", static_shared_memory2);
    printf("dynamic_shared_memory = %p\n", dynamic_shared_memory); // 动态共享变量，无论定义多少个，地址都一样
    printf("dynamic_shared_memory2 = %p\n", dynamic_shared_memory2);

    if (blockIdx.x == 0 && threadIdx.x == 0) // 第一个thread
        printf("Run kernel.\n");
}

/////////////////////demo2//////////////////////////////////
/*
demo2 主要是为了演示的是如何给 共享变量进行赋值
 */
// 定义共享变量，但是不能给初始值，必须由线程或者其他方式赋值
__shared__ int shared_value1;

__global__ void demo2_kernel()
{

    __shared__ int shared_value2;
    if (threadIdx.x == 0)
    {

        // 在线程索引为0的时候，为shared value赋初始值
        if (blockIdx.x == 0)
        {
            shared_value1 = 123;
            shared_value2 = 55;
        }
        else
        {
            shared_value1 = 331;
            shared_value2 = 8;
        }
    }

    // 等待block内的所有线程执行到这一步
    __syncthreads();

    printf("%d.%d. shared_value1 = %d[%p], shared_value2 = %d[%p]\n",
           blockIdx.x, threadIdx.x,
           shared_value1, &shared_value1,
           shared_value2, &shared_value2);
}

void launch()
{

    demo1_kernel<<<1, 1, 12, nullptr>>>();
    demo2_kernel<<<2, 5, 0, nullptr>>>();
}